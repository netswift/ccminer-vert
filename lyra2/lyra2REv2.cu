extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"


static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];
static  uint64_t *d_hash2[MAX_GPUS];

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_hash_80(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void Keccak256_cpu_hash_32(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void skeinCube256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);


extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, uint32_t tpb);
extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t* matrix);

extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t target);

extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash);

extern "C" void lyra2v2_hash(void *state, const void *input)
{
	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);


	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 4, 4);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);


	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2v2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = 256 * 256 * 4;
	uint32_t tpb = 32;
	//bool mergeblakekeccak = false;
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
	if (!opt_eco_mode)
	{
		if (strstr(props.name, "TITAN X"))
		{
			tpb = 32;
			intensity = 256 * 256 * 32;
		}
		else if (strstr(props.name, "970"))
		{
			tpb = 32;
			intensity = 256 * 256 * 32;
		}
		else if (strstr(props.name, "980 Ti"))
		{
			tpb = 32;
			intensity = 256 * 256 * 32;
		}
		else if (strstr(props.name, "980"))
		{
			tpb = 32;
			intensity = 256 * 256 * 32;
		}
		else if (strstr(props.name, "750 Ti"))
		{
			intensity = 256 * 256 * 4;
			tpb = 32;
			//mergeblakekeccak = true;
		}
		else if (strstr(props.name, "750"))
		{
			intensity = 256 * 256 * 4;
			tpb = 32;
			//mergeblakekeccak = true;
		}
		else if (strstr(props.name, "960"))
		{
			tpb = 32;
			intensity = 256 * 256 * 8;
		}
		else if (strstr(props.name, "950"))
		{
			intensity = 256 * 256 * 8;
			tpb = 32;
		}
	}
	else
	{
		if (strstr(props.name, "TITAN X"))
		{
			tpb = 32;
			intensity = 256 * 256 * 4;
		}
		else if (strstr(props.name, "970"))
		{
			tpb = 32;
			intensity = 256 * 256 * 4;
		}
		else if (strstr(props.name, "980 Ti"))
		{
			tpb = 32;
			intensity = 256 * 256 * 4;
		}
		else if (strstr(props.name, "980"))
		{
			tpb = 32;
			intensity = 256 * 256 * 4;
		}
		else if (strstr(props.name, "750 Ti"))
		{
			intensity = 256 * 256 / 2;
			tpb = 32;
			//mergeblakekeccak = true;
		}
		else if (strstr(props.name, "750"))
		{
			intensity = 256 * 256 / 2;
			tpb = 32;
			//mergeblakekeccak = true;
		}
		else if (strstr(props.name, "960"))
		{
			tpb = 32;
			intensity = 256 * 256 * 1;
		}
		else if (strstr(props.name, "950"))
		{
			intensity = 256 * 256 * 1;
			tpb = 32;
		}
	}
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		//keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], 4 * sizeof(uint64_t) * 4 * throughput + 128));
		d_hash2[thr_id] = (uint64_t*)(((uint64_t)d_hash2[thr_id] + 127)&~127);
		lyra2v2_cpu_init(thr_id, throughput, d_hash2[thr_id]);
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * throughput+128));
		d_hash[thr_id] = (uint64_t*)(((uint64_t)d_hash[thr_id] + 127)&~127);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);

	do {
		//applog(LOG_WARNING, "GPU #%d: Loop Start! ptarget = %8x%8x%8x%8x%8x%8x%8x%8x", thr_id, ptarget[7], ptarget[6], ptarget[5], ptarget[4], ptarget[3], ptarget[2], ptarget[1], ptarget[0] );
		uint32_t foundNonce[2] = { 0, 0 };

		//		if (mergeblakekeccak)
		//		{
		blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);

		/*		}
		else
		{
		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		}
		*/

		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], tpb);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce, ptarget[7]);

		//		foundNonce[0] = 0xffffffff;
		if (foundNonce[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			lyra2v2_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0xffffffff)
				{
					pdata[21] = foundNonce[1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundNonce[1], vhash64[7], Htarg);
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce % 08x", thr_id, foundNonce[0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;

	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	
	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
